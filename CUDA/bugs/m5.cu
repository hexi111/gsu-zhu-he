
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

#define SIZE 512

// This example is adapted from an example in Nvidia CUDA C Programming Guide 4.0

__global__ void demo(int * input,int* output) {
	int tid = threadIdx.x;
	int ref1 = input[tid];
	//These two syncthreads call can make sure memoey coherence. 
	//__syncthreads();
	input[tid + 1] = 2;
	//__syncthreads();
	int ref2 = input[tid];
	output[tid] = ref1 * ref2;
}
int main(int argc , char **argv){
	hipError_t err;
	int * input;
	err=hipMalloc((void**)&input,(SIZE+1)*sizeof(int));
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	int * output;
	err=hipMalloc((void**)&output,SIZE*sizeof(int));
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	int * temp;
	temp=(int *)malloc((SIZE+1)*sizeof(int));
	int i;
	for(i=0;i<(SIZE+1);i++){
		temp[i]=1;
	}
	err=hipMemcpy( input, temp, sizeof(int)*(SIZE+1), hipMemcpyHostToDevice);
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	dim3 dimGrid(1,1);
	dim3 dimBlock(SIZE,1);
	demo<<<dimGrid,dimBlock>>>(input,output);
	err=hipMemcpy( temp, output, sizeof(int)*(SIZE), hipMemcpyDeviceToHost);
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	for(i=0;i<SIZE;i++){
		printf("%dth element is %d\n",i,temp[i]);
	}
	free(temp);
	err=hipFree(input);
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	err=hipFree(output);
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
    return 0;
}
