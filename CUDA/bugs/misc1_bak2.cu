
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

#define NUMBER 100

__global__ void demo(int *arr){
	int tx=threadIdx.x;
	if(tx>NUMBER){
		return;
	}
	arr[tx]+=tx;
	__syncthreads();	
}
int main(int argc , char **argv){
	int * arr;
	hipError_t err;
	err=hipMalloc((void**)&arr,NUMBER*sizeof(int));
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}	
	dim3 dimGrid(1,1);
	dim3 dimBlock(512,1);
	demo<<<dimGrid,dimBlock>>>(arr);
	err=hipFree(arr);
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
    return 0;
}
