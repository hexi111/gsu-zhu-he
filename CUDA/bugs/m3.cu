
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

int main(int argc , char **argv){
	size_t mem_tot = 0;
	size_t mem_free = 0;
	hipMemGetInfo (&mem_free, &mem_tot);
	printf("total memory %ld bytes, free memory %ld bytes\n", (long) mem_tot, (long) mem_free );
	int * p;
	long size=1024*1024;
	size*=1024*2;
	printf("size=%ld\n",size);
	hipError_t err;
	err=hipMalloc((void**)&p,size*sizeof(int));;	
	// Need to catch the exception from GPU side.
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	hipMemGetInfo (&mem_free, &mem_tot);
	printf("total memory %ld bytes, free memory %ld bytes\n", (long) mem_tot, (long) mem_free );
}
