
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

#define SIZE 1000

__global__ void demo(int * p){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	int thid = tx+bx*blockDim.x;
	// Some of the threads try to access memory out of array boundary. 
	// The program may not get any error message, but will pose a potential bug. 
	p[thid]=thid+p[thid];
}
int main(int argc , char **argv){
	int * p_cpu;
	int * p_gpu;
	
	hipError_t err;
	err=hipMalloc((void**)&p_gpu,SIZE*sizeof(int));
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	p_cpu=(int *)malloc(SIZE*sizeof(int));

	int i;
	for(i=0;i<SIZE;i++){
		p_cpu[i]=1;	
	}
	
	err=hipMemcpy( p_gpu, p_cpu, sizeof(int)*SIZE, hipMemcpyHostToDevice);
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	dim3 dimGrid((SIZE-1)/512+1,1);
	dim3 dimBlock(512,1);
	demo<<<dimGrid,dimBlock>>>(p_gpu);
	free(p_cpu);
	err=hipFree(p_gpu);
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
    return 0;
}
