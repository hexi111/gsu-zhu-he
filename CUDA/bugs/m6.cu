
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

#define SIZE 10

int main(int argc , char **argv){

	int * p;
	hipError_t err;
	// Should be cudaMalloc((void**)&p,SIZE*sizeof(int))
	err=hipMalloc((void**)&p,SIZE);
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	hipFree(p);
    return 0;
}
