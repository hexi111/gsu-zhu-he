
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

#define NUMBER 100

__global__ void demo(int *arr){
    if(threadIdx.x<NUMBER){
      for(int i=0;i<NUMBER;i++){
        arr[threadIdx.x]= arr[threadIdx.x]+threadIdx.x;
        __syncthreads();
       }
    }
}
int main(int argc , char **argv){
	int * arr;
	hipError_t err;
	err=hipMalloc((void**)&arr,NUMBER*sizeof(int));
	if( err != hipSuccess){
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}	
	dim3 dimGrid(1,1);
	dim3 dimBlock(512,1);
	demo<<<dimGrid,dimBlock>>>(arr);
	err=hipFree(arr);
	if( err != hipSuccess){
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
    return 0;
}
