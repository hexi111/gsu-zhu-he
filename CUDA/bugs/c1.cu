
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

#define SIZE 2000

__global__ void demo(int * p){
	int tx=threadIdx.x;
	printf("tx=%d\n",tx);
	p[tx]=tx;
}
int main(int argc , char **argv){
	int * p;
	hipError_t err;
	err=hipMalloc((void**)&p,SIZE*sizeof(int));
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	dim3 dimGrid(1,1);
	dim3 dimBlock(SIZE,1);
	// Configuration too many thread in a thread block.
	demo<<<dimGrid,dimBlock>>>(p);
	err=hipFree(p);
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}   
	printf("tx\n");
 
	return 0;
}
