
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

#define NUMBER 512

__global__ void reduction(int *arr,int num){
	int tx=threadIdx.x;
	int round=1;
	arr[tx]=1;
	__syncthreads();	
	while(round<NUMBER){
		if((tx%round==0)&&((tx+round)<NUMBER)){
			arr[tx]=arr[tx+round]+arr[tx];
			__syncthreads();
		}
		round=round<<1;
		//__syncthreads();
	}
	if(tx==0){
		printf("Sum of the array is %d\n",arr[0]);
	}
}
int main(int argc , char **argv){
	int * arr;
	hipError_t err;
	err=hipMalloc((void**)&arr,NUMBER*sizeof(int));
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}	
	dim3 dimGrid(1,1);
	dim3 dimBlock(NUMBER,1);
	reduction<<<dimGrid,dimBlock>>>(arr,NUMBER);
	err=hipFree(arr);
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
    return 0;
}
