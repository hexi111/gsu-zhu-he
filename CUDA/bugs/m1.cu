
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

#define SIZE 10

__global__ void demo(int * p){
	int tx=threadIdx.x;
	// Trying to access variables allocated on main memory in a kernel function is illegal. 
	p[tx]=tx+p[tx];
}
int main(int argc , char **argv){
	int * p;
	p=(int *)malloc(10*sizeof(int));
	int i;
	for(i=0;i<SIZE;i++){
		p[i]=1;	
	}
	dim3 dimGrid(1,1);
	dim3 dimBlock(SIZE,1);
	demo<<<dimGrid,dimBlock>>>(p);
	for(i=0;i<SIZE;i++){
		printf("p[%d]=%d\n",i,p[i]);		
	}
	free(p);
    return 0;
}
