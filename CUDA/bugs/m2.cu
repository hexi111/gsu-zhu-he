
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

#define SIZE 10

int main(int argc , char **argv){

	int * p;
	hipError_t err;
	err=hipMalloc((void**)&p,SIZE*sizeof(int));
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	int i;
	for(i=0;i<SIZE;i++){
		//Accessing variables allocated on global memory in host function causes Segmentation fault.
		p[i]=1;	
	}
	hipFree(p);
    return 0;
}
