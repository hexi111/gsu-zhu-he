
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

#define THREADSIZE 100
#define BLOCKSIZE 65536

__global__ void demo(int * p){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	int thid = tx+bx*blockDim.x;
	p[thid]=thid;
	if(thid<10){
		printf("%d elements is %d\n",thid,p[thid]);
	}
}
int main(int argc , char **argv){
	int * p;
	hipError_t err;
	err=hipMalloc((void**)&p,THREADSIZE*BLOCKSIZE*sizeof(int));
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}
	dim3 dimGrid(BLOCKSIZE,1);
	dim3 dimBlock(THREADSIZE,1);
	// Configuration is not correct.
	// x, y, z dimension of thread blocks should not exceed 65535 in compute capability 2.0
	demo<<<dimGrid,dimBlock>>>(p);
	err=hipFree(p);
	if( err != hipSuccess)
	{
    	printf("CUDA error: %s\n", hipGetErrorString(err));
     	exit(-1);
	}   
	printf("tx\n");
 
	return 0;
}
